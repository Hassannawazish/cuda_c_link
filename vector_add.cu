#include <hip/hip_runtime.h>

__global__ void vector_add_kernel(float *out, float *a, float *b, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        out[i] = a[i] + b[i];
}

// Wrapper function for the __global__ call
void vector_add(float **out, float **a, float **b, int n) {
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // Allocate device memory
    hipMallocManaged(a, sizeof(float) * n);
    hipMallocManaged(b, sizeof(float) * n);
    hipMallocManaged(out, sizeof(float) * n);

    for(int i = 0; i < n; i++){
        (*a)[i] = i;
        (*b)[i] = i;
    }

    vector_add_kernel<<<numBlocks, blockSize>>>(*out, *a, *b, n);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
}

void deallocate(float *a, float *b, float *out){
    // Free device memory
    hipFree(a);
    hipFree(b);
    hipFree(out);
}
